#include "utils/ppm.hpp"
#include "test.hpp"
#include <iostream>
#include <string>
#include <iomanip>
#include <cstdio> // For sscanf

void printUsage() 
{
    std::cerr << "Usage: " << std::endl
              << " \t -f <F>: <F> image file name"
              << std::endl << std::endl;
    exit(EXIT_FAILURE);
}

int main(int argc, char **argv) 
{   
    char fileName[2048];

    // Parse program arguments
    if (argc == 1) 
    {
        std::cerr << "Please give a file..." << std::endl;
        printUsage();
    }

    for (int i = 1; i < argc; ++i) 
    {
        if (!strcmp(argv[i], "-f")) 
        {
            if (sscanf(argv[++i], "%2047s", fileName) != 1)
                printUsage();
        }
        else
            printUsage();
    }

    // ================================================================================================================
    // Get input image
    std::cout << "Loading image: " << fileName << std::endl;
    const los::Heightmap input(fileName);
    const int width = input.getWidth();
    const int height = input.getHeight();

    std::cout << "Image has " << width << " x " << height << " pixels" << std::endl;

    std::string baseSaveName = fileName;
    baseSaveName.erase(baseSaveName.end() - 4, baseSaveName.end()); // erase .ppm

    // Create 3 output images
    los::Heightmap outCPU(width, height);

    // ================================================================================================================

    // CPU sequential
    std::cout << "============================================" << std::endl;
    std::cout << "         Sequential version on CPU          " << std::endl;
    std::cout << "============================================" << std::endl;

    Point center(245, 497);
	
    const float timeCPU = naive_viewtestCPU(input, center, outCPU);

    std::string cpuName = baseSaveName + "resultat.ppm";
    outCPU.saveTo(cpuName.c_str());

    std::cout << "-> Done : " << timeCPU << " ms" << std::endl << std::endl;

    // ================================================================================================================

    // GPU CUDA

    return EXIT_SUCCESS;
}
